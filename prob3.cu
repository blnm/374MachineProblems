#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define MATRIX_DIM 500
#define MATRIX_SIZE (MATRIX_DIM * MATRIX_DIM)
#define BLOCK_WIDTH 16

void mulWithCuda(float *p, const float *m, const float *n);
void testTransferTime(float *m, float *n);

// calculate one element with one thread
__global__ void matrixMultiplication(float *P, float *M, float *N)
{
	// calculate row, col index
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	if (row < MATRIX_DIM && col < MATRIX_DIM)
	{
		float p_val = 0;

		// do multiplication here

		P[row*MATRIX_DIM + col] = p_val; // M[ind] + N[ind];
	}
	__syncthreads();
}

void verifyGPUsoln(const float *GPU_C, const float *A, const float *B)
{
	bool passed = true;
	for (int i = 0; i < MATRIX_SIZE; i++)
	{
		if (GPU_C[i] != A[i] + B[i])
		{
			passed = false;
			break;
		}
	}

	if (passed)	printf("TEST PASSED\n");
	else		printf("TEST FAILED\n");
}


int main()
{
	float *a = (float *)malloc(MATRIX_SIZE * sizeof(float)); // yeah, there ain't enough room on the
	float *b = (float *)malloc(MATRIX_SIZE * sizeof(float)); // stack for 3 5000x5000 matricies
	float *c = (float *)malloc(MATRIX_SIZE * sizeof(float)); // you can mess with VS's settings but I'd rather not.

	for (int i = 0; i < MATRIX_SIZE; i++)
	{	// value between 0 and 10, one decimal place
		a[i] = rand() % 100 / 10.0;
		b[i] = rand() % 100 / 10.0;
	}

	testTransferTime(a, b);

	//mulWithCuda(p, m, n);

	//verifyGPUsoln(p, m, n);

	free(a);
	free(b);
	free(c);

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
void mulWithCuda(float *p, const float *m, const float *n)
{
	float *dev_m = 0;
	float *dev_n = 0;
	float *dev_p = 0;

	hipError_t malloc_test;

	// Allocate GPU buffers for three vectors (two input, one output)
	malloc_test = hipMalloc((void**)&dev_p, MATRIX_SIZE * sizeof(float));	// p
	if (malloc_test != hipSuccess) printf("error allocating mem for dev_p\n");

	malloc_test = hipMalloc((void**)&dev_m, MATRIX_SIZE * sizeof(float));	// m
	if (malloc_test != hipSuccess) printf("error allocating mem for dev_m\n");

	malloc_test = hipMalloc((void**)&dev_n, MATRIX_SIZE * sizeof(float));	// n
	if (malloc_test != hipSuccess) printf("error allocating mem for dev_n\n");

	// Copy input vectors from host memory to GPU.
	hipMemcpy(dev_m, m, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_n, n, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);


	// create event-based timers
	hipEvent_t start, stop;
	float gpu_time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// generall error for checking errors during GPU processing
	hipError_t mulErr;


	// create block/thread dims with add-by-element as default
	int numBlocks = MATRIX_DIM / BLOCK_WIDTH;
	if (MATRIX_DIM % BLOCK_WIDTH) numBlocks++;
	dim3 grid(numBlocks, numBlocks);
	dim3 block(BLOCK_WIDTH, BLOCK_WIDTH);

	hipEventRecord(start, 0); // start timer

	matrixMultiplication <<<grid, block >>>(dev_p, dev_m, dev_n);
	mulErr = hipGetLastError();
	if (mulErr != hipSuccess) printf("Error during addition: %s", hipGetErrorString(addErr));


	hipEventRecord(stop, 0);	// end timer and display results
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_time, start, stop);

	printf("time taken (ms):\t%.2f\n", gpu_time); //TODO - add output of block size to make excel easier


	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
	hipError_t cpyErr = hipMemcpy(p, dev_p, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost);

	if (cpyErr != hipSuccess) printf("error copying dev_p to host\n");

	hipError_t freeErr;
	freeErr = hipFree(dev_p);
	if (freeErr != hipSuccess) printf("error freeing dev_p\n");	// p

	freeErr = hipFree(dev_m);
	if (freeErr != hipSuccess) printf("error freeing dev_m\n");	// m

	freeErr = hipFree(dev_n);
	if (freeErr != hipSuccess) printf("error freeing dev_n\n");	// n
}


void testTransferTime(float *m, float *n)
{
	float *dev_m = 0;
	float *dev_n = 0;

	hipError_t malloc_test;

	// Allocate GPU buffers for two vectors
	malloc_test = hipMalloc((void**)&dev_m, MATRIX_SIZE * sizeof(float));	// m
	if (malloc_test != hipSuccess) printf("error allocating mem for dev_m\n");

	malloc_test = hipMalloc((void**)&dev_n, MATRIX_SIZE * sizeof(float));	// n
	if (malloc_test != hipSuccess) printf("error allocating mem for dev_n\n");

	// create event-based timers
	hipEvent_t start, stop;
	float gpu_time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0); // start timer

							   // Copy from host memory to GPU.
	hipMemcpy(dev_m, m, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_n, n, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);

	hipEventRecord(stop, 0);	// end timer and display results
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_time, start, stop);
	printf("copy host to device (ms):\t%.2f\n", gpu_time);

	hipEventRecord(start, 0); // start timer

							   // Copy from GPU buffer to host memory.
	hipMemcpy(m, dev_m, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(n, dev_n, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);	// end timer and display results
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_time, start, stop);
	printf("copy device to host (ms):\t%.2f\n", gpu_time);

	hipError_t freeErr;
	freeErr = hipFree(dev_m);
	if (freeErr != hipSuccess) printf("error freeing dev_p\n");	// m

	freeErr = hipFree(dev_n);
	if (freeErr != hipSuccess) printf("error freeing dev_m\n");	// n
}