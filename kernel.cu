#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>

__global__ void increment_kernel(int *g_data, int inc_value)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	g_data[idx] = g_data[idx] + inc_value;
}

int correct_output(int *data, const int n, const int x)
{
	for (int i = 0; i < n; i++)
		if (data[i] != x)
			return 0;
	return 1;
}

int main()
{
	hipDeviceProp_t deviceProps;

	// get dev name
	hipGetDeviceProperties(&deviceProps, 0);
	printf("CUDA device [%s]\n", deviceProps.name);

	int n = 16 * 1024 * 1024;
	int nbytes = n * sizeof(int);
	int value = 26;

	// alloc host mem
	int *a = 0;
	hipHostMalloc((void**)&a, nbytes, hipHostMallocDefault);
	// memset(a, 0, nbytes); // commented out in source code, included just in case

	// alloc device mem
	int *d_a = 0;
	hipMalloc((void**)&d_a, nbytes);
	hipMemset(d_a, 255, nbytes);

	// set kernel launch config
	dim3 threads = dim3(512, 1);
	dim3 blocks = dim3(n / threads.x, 1);

	// create cuda event handles
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipDeviceSynchronize();
	float gpu_time = 0.0f;

	// asynchronously issue work to GPU (all to stream 0)
	hipEventRecord(start, 0);
	hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);

	increment_kernel<<<blocks, threads, 0, 0 >>>(d_a, value);

	hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
	hipEventRecord(stop, 0);

	// have CPU do some work while waiting for GPU to finish
	unsigned long int counter = 0;
	while (hipEventQuery(stop) == hipErrorNotReady)
	{
		counter++; // indicates that the CPU is running asynchronously while GPU is executing
	}

	hipEventSynchronize(stop); // stop is updated here
	hipEventElapsedTime(&gpu_time, start, stop); // time difference between start and stop

	// print the GPU times
	printf("time spent executing by the GPU: %.2f\n", gpu_time);
	printf("CPU executed %d iterations while waiting for GPU to finish\n", counter);

	// check the output for correctness
	bool bFinalResults = (bool)correct_output(a, n, value);

	// release resources
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipHostFree(a);
	hipFree(d_a);
	hipDeviceReset();

	return 0;
}